#include "hip/hip_runtime.h"
// @file spp_gpu.cu
// @brief  SPP block implementation (GPU)
// @author Hakan Bilen

#include "spp.hpp"
#include "bits/datamex.hpp"
#include "bits/datacu.hpp"
#include "matrix.h"

#include <float.h>
#include <sm_20_atomic_functions.h>
#include <cmath>
#include <stdio.h>

/* ---------------------------------------------------------------- */
/*                                              spp_max_forward */
/* ---------------------------------------------------------------- */
template<typename T> __global__ void
spp_avg_kernel
(T* pooled,
 const T* data,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const float * levels,
 const int numROIs,
 const float * ROIs)
{
  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");
  mxAssert(numTotBins>0,"");


  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;


  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {

    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no

    mxAssert(pr<numROIs,"");

    int roi_image   = ROIs[5 * pr + 0];
    mxAssert(roi_image<size,"");

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }

    mxAssert(roi_start_w>=0,"");
    mxAssert(roi_start_h>=0,"");
    mxAssert(roi_end_w<width,"");
    mxAssert(roi_end_h<height,"");

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }
    
    mxAssert(pb>=0,"");
    mxAssert(pLevel>=0,"");
    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const float bin_size_h = static_cast<float>(roi_height)
        / static_cast<float>(pooledHeight);
    const float bin_size_w = static_cast<float>(roi_width)
        / static_cast<float>(pooledWidth);


    mxAssert(ph>-1 && pw>-1,"");

    int hstart = static_cast<int>(floor(static_cast<float>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<float>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<float>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<float>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);

    int offset_data = (roi_image * depth + pc) * (width*height);
    mxAssert(offset_data<width*height*depth*size,"");

    data += offset_data;
    T bestValue = 0;
    const float coef = 1.f / (float)((wend-wstart) * (hend-hstart));
    for (int w = wstart; w < wend; ++w) {
      for (int h = hstart; h < hend; ++h) {
        int index = w * height + h ;
        bestValue += data[index] * coef;
      }
    }
    pooled[pooledIndex] = bestValue ;
  }
}

template<typename T> __global__ void
spp_max_kernel
(T* pooled,
 const T* data,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const float * levels,
 const int numROIs,
 const float * ROIs)
{
  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");
  mxAssert(numTotBins>0,"");


  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;


  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {

    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no
    mxAssert(pr<numROIs,"");

    int roi_image   = ROIs[5 * pr + 0];
    mxAssert(roi_image<size,"");

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }

    mxAssert(roi_start_w>=0,"");
    mxAssert(roi_start_h>=0,"");
    mxAssert(roi_end_w<width,"");
    mxAssert(roi_end_h<height,"");

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }
    
    mxAssert(pb>=0,"");
    mxAssert(pLevel>=0,"");
    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const float bin_size_h = static_cast<float>(roi_height)
        / static_cast<float>(pooledHeight);
    const float bin_size_w = static_cast<float>(roi_width)
        / static_cast<float>(pooledWidth);

//    free(numBins);

    mxAssert(ph>-1 && pw>-1,"");

    int hstart = static_cast<int>(floor(static_cast<float>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<float>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<float>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<float>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int offset_data = (roi_image * depth + pc) * (width*height);
    mxAssert(offset_data<width*height*depth*size,"");

    data += offset_data;
    T bestValue = is_empty ? 0 : data[wstart * height + hstart];
    for (int w = wstart; w < wend; ++w) {
      for (int h = hstart; h < hend; ++h) {
        int index = w * height + h ;
        bestValue = max(bestValue, data[index]) ;
      }
    }
    pooled[pooledIndex] = bestValue ;

  }
}

template<> vl::Error
vl::impl::spp_avg_forward<vl::GPU, float>(float* pooled,
                                          float const* data,
                                          size_t height, size_t width, size_t depth, size_t size,
                                          size_t numTotBins,
                                          size_t numLevels, const float * levels,
                                          size_t numROIs, const float * ROIs)
{
  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");

  int pooledVolume = numTotBins * depth * numROIs;

  spp_avg_kernel<float><<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS),
      VL_CUDA_NUM_THREADS >>>(pooled, data,
                              height, width, depth, size,
                              numTotBins,
                              numLevels, levels,
                              numROIs, ROIs);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

template<> vl::Error
vl::impl::spp_max_forward<vl::GPU, float>(float* pooled,
                                          float const* data,
                                          size_t height, size_t width, size_t depth, size_t size,
                                          size_t numTotBins,
                                          size_t numLevels, const float * levels,
                                          size_t numROIs, const float * ROIs)
{
  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");

  int pooledVolume = numTotBins * depth * numROIs;

  spp_max_kernel<float><<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS),
      VL_CUDA_NUM_THREADS >>>(pooled, data,
                              height, width, depth, size,
                              numTotBins,
                              numLevels, levels,
                              numROIs, ROIs);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}
/* ---------------------------------------------------------------- */
/*                                              spp_max_backward */
/* ---------------------------------------------------------------- */
template<typename T> __global__ void
spp_max_backward_kernel
(T* derData,
 const T* data,
 const T* derPooled,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const float * levels,
 const int numROIs,
 const float * ROIs)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;

  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");

  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {


    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no
    mxAssert(pr<numROIs,"");

    int roi_image   = ROIs[5 * pr + 0];
    mxAssert(roi_image<size,"");

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }
    mxAssert(roi_start_w>=0,"");
    mxAssert(roi_start_h>=0,"");
    mxAssert(roi_end_w<width,"");
    mxAssert(roi_end_h<height,"");

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }
    
    mxAssert(pb>=0,"");
    mxAssert(pLevel>=0,"");
    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const float bin_size_h = static_cast<float>(roi_height)
        / static_cast<float>(pooledHeight);
    const float bin_size_w = static_cast<float>(roi_width)
        / static_cast<float>(pooledWidth);


    mxAssert(ph>-1 && pw>-1,"");

    int hstart = static_cast<int>(floor(static_cast<float>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<float>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<float>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<float>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    data += (roi_image * depth + pc) * (width*height);
    derData += (roi_image * depth + pc) * (width*height);

    int bestIndex = wstart * height + hstart;
    T bestValue = is_empty ? 0 : data[bestIndex];
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int index = w * height + h ;
        T value = data[index] ;
        if (value > bestValue) {
          bestValue = value ;
          bestIndex = index ;
        }
      }
    }

    /*
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    atomicAdd(derData + bestIndex, derPooled[pooledIndex]) ;
  }
}


/* ---------------------------------------------------------------- */
/*                                              spp_avg_backward */
/* ---------------------------------------------------------------- */
template<typename T> __global__ void
spp_avg_backward_kernel
(T* derData,
 const T* data,
 const T* derPooled,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const float * levels,
 const int numROIs,
 const float * ROIs)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;

  mxAssert(numLevels>0,"numLevels>0");
  mxAssert(numROIs>0,"numROIs>0");

  //  int numTotBins = 0;
  //  numTotBins = numBins[numLevels];

  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {

    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no
    mxAssert(pr<numROIs,"");

    int roi_image   = ROIs[5 * pr + 0];
    mxAssert(roi_image<size,"");

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }
    mxAssert(roi_start_w>=0,"");
    mxAssert(roi_start_h>=0,"");
    mxAssert(roi_end_w<width,"");
    mxAssert(roi_end_h<height,"");

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }
    
    mxAssert(pb>=0,"");
    mxAssert(pLevel>=0,"");
    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const float bin_size_h = static_cast<float>(roi_height)
        / static_cast<float>(pooledHeight);
    const float bin_size_w = static_cast<float>(roi_width)
        / static_cast<float>(pooledWidth);


    mxAssert(ph>-1 && pw>-1,"");

    int hstart = static_cast<int>(floor(static_cast<float>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<float>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<float>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<float>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);

    data += (roi_image * depth + pc) * (width*height);
    derData += (roi_image * depth + pc) * (width*height);

    const float coef = 1.f / (float)((wend-wstart)*(hend-hstart));
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int index = w * height + h ;
      /*
       This is bad, but required to eliminate a race condition when writing
       to bottom_diff.
       Caffe goes the other way around, but requires remembering the layer
       output, or the maximal indexes.
       atomicAdd(add, val)
       */
        atomicAdd(derData + index, derPooled[pooledIndex] * coef) ;
      }
    }


  }
}

template<> vl::Error
vl::impl::spp_max_backward<vl::GPU, float>(float* derData,
                                           float const* data,
                                           float const* derPooled,
                                           size_t height, size_t width, size_t depth, size_t size,
                                           size_t numTotBins,
                                           size_t numLevels, const float * levels,
                                           size_t numROIs, const float * ROIs)
{
  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");

  int pooledVolume = numTotBins * depth * numROIs;

  spp_max_backward_kernel<float>
      <<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
                                                                                  (derData, data, derPooled,
                                                                                   height, width, depth, size,
                                                                                   numTotBins,
                                                                                   numLevels, levels,
                                                                                   numROIs, ROIs);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

template<> vl::Error
vl::impl::spp_avg_backward<vl::GPU, float>(float* derData,
                                           float const* data,
                                           float const* derPooled,
                                           size_t height, size_t width, size_t depth, size_t size,
                                           size_t numTotBins,
                                           size_t numLevels, const float * levels,
                                           size_t numROIs, const float * ROIs)
{
  mxAssert(numLevels>0,"");
  mxAssert(numROIs>0,"");

  int pooledVolume = numTotBins * depth * numROIs;

  spp_avg_backward_kernel<float>
      <<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
                                                                                  (derData, data, derPooled,
                                                                                   height, width, depth, size,
                                                                                   numTotBins,
                                                                                   numLevels, levels,
                                                                                   numROIs, ROIs);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}
