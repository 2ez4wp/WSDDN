#include "hip/hip_runtime.h"
// @file spp_cpu.cpp
// @brief SPP block implementation (GPU)
// @author Hakan Bilen 

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "nnspp.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* option codes */
enum {
    opt_numbins=0,
    opt_method,
    opt_verbose,
} ;

/* options */
vlmxOption  options [] = {
    {"NumBins",          1,   opt_numbins },
    {"Method",           1,   opt_method },
    {"Verbose",          0,   opt_verbose },
    {0,                  0,   0           }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
    context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
    IN_DATA = 0, IN_LEVELS, IN_ROIS, IN_DEROUTPUT, IN_END
} ;

enum {
    OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
    int numLevels = 0;
    int numROIs = 0;
    int numTotBins = 0;
    // 0 -> max, 1 -> avg
    int method = 0;

    bool backMode = false ;

    int verbosity = 0 ;
    int opt ;
    int next = IN_END ;
    mxArray const *optarg ;

    /* -------------------------------------------------------------- */
    /*                                            Check the arguments */
    /* -------------------------------------------------------------- */

    mexAtExit(atExit) ;

    if (nin < 3) {
        mexErrMsgTxt("The arguments are less than three.") ;
    }

    if (nin > 3 && vlmxIsString(in[3],-1)) {
        next = 3 ;
        backMode = 0 ;
    } else {
        backMode = (nin >= 4) ;
    }

    while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
        switch (opt) {
        case opt_verbose :
            ++ verbosity ;
            break ;
        case opt_numbins :
            if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
                mexErrMsgTxt("NUMBINS is not a plain matrix.") ;
            }
            numTotBins = (int)mxGetPr(optarg)[0] ;
            break ;
        case opt_method :
            if (!vlmxIsString(optarg,-1)) {
                vlmxError(vlmxErrInvalidArgument, "METHOD is not a string.") ;
            }
            if (vlmxIsEqualToStringI(optarg, "max")) {
                method = 0 ;
            } else if (vlmxIsEqualToStringI(optarg, "avg")) {
                method = 1 ;
            } else {
                vlmxError(vlmxErrInvalidArgument, "METHOD is not a supported method.") ;
            }
        default:
            break ;
        }
    }


    vl::MexTensor data(context) ;
    vl::MexTensor derOutput(context) ;

    vl::MexTensor dROIs(context) ;
    vl::MexTensor dlevels(context) ;

    // load pyramid levels and rois
    dlevels.init(in[IN_LEVELS]);
    dROIs.init(in[IN_ROIS]);


    if (verbosity > 0) {
        mexPrintf("vl_nnspp.cu: numTotBins %d\n",numTotBins);
        mexPrintf("levels %d %d %d %d\n",dlevels.getWidth(),dlevels.getHeight(),dlevels.getDepth(),dlevels.getSize());
        mexPrintf("dROIs %d %d %d %d\n",dROIs.getWidth(),dROIs.getHeight(),dROIs.getDepth(),dROIs.getSize());
        mexPrintf("vl_nnspp.cu: numTotBins %d\n",numTotBins);
    }
    //  mexErrMsgTxt("levels and dROIs are not in right size.") ;

    data.init(in[IN_DATA]) ;
    if (backMode) { derOutput.init(in[IN_DEROUTPUT]) ; }

    if (backMode && ! vl::areCompatible(data, derOutput)) {
        mexErrMsgTxt("DATA and DEROUTPUT are not both CPU or GPU arrays.") ;
    }

    numLevels = dlevels.getNumElements();
    if (numLevels<=0) {
        mexErrMsgTxt("LEVELS has zero elements.") ;
    }

    numROIs = dROIs.getNumElements();


    if (numROIs % 5 != 0) {
        mexErrMsgTxt("Wrong number of elements in ROIS.") ;
    }

    if (numROIs<=0) {
        mexErrMsgTxt("ROIs has zero elements.") ;
    }

    numROIs /= 5;


    if(numTotBins<=0) {
        mexPrintf("numTotBins %d\n",numTotBins);
        mexErrMsgTxt("numTotBins is wrong.") ;
    }

    if (verbosity > 0) {
        mexPrintf("numTotBins %d depth %d numROIs %d\n",numTotBins,data.getDepth(),numROIs);
    }
    /* Get the output geometry */
    vl::TensorShape outputShape(1, numTotBins,
                                data.getDepth(),
                                numROIs) ;

    vl::TensorShape dataShape = data.getShape();

    if(dataShape.getNumDimensions()<4) {
        dataShape.reshape(4);
    }

    /* Create output buffers */
    vl::Device deviceType = data.getDeviceType() ;
    vl::Type dataType = data.getDataType() ;
    vl::MexTensor output(context) ;
    vl::MexTensor derData(context) ;

    if (verbosity > 0) {
        vl::print("vl_nnspp: data: ", data) ;
        if (backMode) {
            vl::print("vl_nnspp: derOutput: ", derOutput) ;
            vl::print("vl_nnspp: derData: ", derData) ;
        } else {
            vl::print("vl_nnspp: output: ", output) ;
        }
    }



    if (!backMode) {
        output.initWithZeros(deviceType, dataType, outputShape) ;
    } else {
        derData.initWithZeros(deviceType, dataType, dataShape) ;
    }

    if (verbosity > 0) {
        mexPrintf("vl_spp: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::GPU) ? "GPU" : "CPU") ;
        mexPrintf("\nvl_spp: method %d numLevels %d; numROIs %d numTotBins %d\n", method, numLevels, numROIs, numTotBins);
    }

    /* -------------------------------------------------------------- */
    /*                                                    Do the work */
    /* -------------------------------------------------------------- */

    vl::Error error ;
    if (!backMode) {
        error = vl::nnspp_forward(context,
                                  output, data,
                                  method,
                                  numTotBins,
                                  dlevels,
                                  dROIs) ;

    } else {
        error = vl::nnspp_backward(context,
                                   derData, data, derOutput,
                                   method,
                                   numTotBins,
                                   dlevels,
                                   dROIs) ;
    }

    /* -------------------------------------------------------------- */
    /*                                                         Finish */
    /* -------------------------------------------------------------- */

    if (error != vl::vlSuccess) {
        mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
    }
    if (backMode) {
        out[OUT_RESULT] = derData.relinquish() ;
    } else {
        out[OUT_RESULT] = output.relinquish() ;
    }

    //  free(levels);
}
