#include "hip/hip_runtime.h"
// @file vl_nnspp.cu
// @brief Spatial Pyramid Pooling MEX wrapper
// @author Hakan Bilen 
// @author Andrea Vedaldi
/*
Copyright (C) 2016- Hakan Bilen and Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

/** this is the mex-wrapper -- entry-point from matlab to cuda */

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnspp.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* option codes */
enum {
    opt_numbins=0,
    opt_method,
    opt_verbose,
} ;

/* options */
vlmxOption  options [] = {
    {"Method",           1,   opt_method },
    {"Verbose",          0,   opt_verbose },
    {0,                  0,   0           }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
    context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
    IN_DATA = 0, IN_LEVELS, IN_ROIS, IN_DEROUTPUT, IN_END
} ;

enum {
    OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
    size_t numLevels = 0;
    size_t numROIs = 0;
    size_t numTotBins = 0;
    // 0 -> max, 1 -> avg
    int method = 0;

    bool backMode = false ;

    int verbosity = 0 ;
    int opt ;
    int next = IN_END ;
    mxArray const *optarg ;

    /* -------------------------------------------------------------- */
    /*                                            Check the arguments */
    /* -------------------------------------------------------------- */

    mexAtExit(atExit) ;

    if (nin < 3) {
        mexErrMsgTxt("The arguments are less than three.") ;
    }

    if (nin > 3 && vlmxIsString(in[3],-1)) {
        next = 3 ;
        backMode = 0 ;
    } else {
        backMode = (nin >= 4) ;
    }

    while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
        switch (opt) {
        case opt_verbose :
            ++ verbosity ;
            break ;
        case opt_method :
            if (!vlmxIsString(optarg,-1)) {
                vlmxError(vlmxErrInvalidArgument, "METHOD is not a string.") ;
            }
            if (vlmxIsEqualToStringI(optarg, "max")) {
                method = 0 ;
            } else if (vlmxIsEqualToStringI(optarg, "avg")) {
                method = 1 ;
            } else {
                vlmxError(vlmxErrInvalidArgument, "METHOD is not a supported method.") ;
            }
        default:
            break ;
        }
    }


    vl::MexTensor data(context) ;
    vl::MexTensor derOutput(context) ;

    vl::MexTensor dROIs(context) ;
    vl::MexTensor pyrLevels(context) ;

    // load pyramid levels and rois
    pyrLevels.init(in[IN_LEVELS]);
    dROIs.init(in[IN_ROIS]);

    size_t elemPL = mxGetNumberOfElements(in[IN_LEVELS]);

    for (size_t i=0;i<elemPL;i++)
      numTotBins += mxGetPr(in[IN_LEVELS])[i];

    if(numTotBins<=0) {
        mexPrintf("numTotBins %d\n",numTotBins);
        mexErrMsgTxt("numTotBins is wrong.") ;
    }


    if (verbosity > 0) {
        mexPrintf("vl_nnspp.cu: numTotBins %d\n",numTotBins);
        mexPrintf("levels %d %d %d %d\n",pyrLevels.getWidth(),pyrLevels.getHeight(),pyrLevels.getDepth(),pyrLevels.getSize());
        mexPrintf("dROIs %d %d %d %d\n",dROIs.getWidth(),dROIs.getHeight(),dROIs.getDepth(),dROIs.getSize());
        mexPrintf("vl_nnspp.cu: numTotBins %d\n",numTotBins);
    }

    data.init(in[IN_DATA]) ;
    if (backMode) { derOutput.init(in[IN_DEROUTPUT]) ; }

    if (backMode && ! vl::areCompatible(data, derOutput)) {
        mexErrMsgTxt("DATA and DEROUTPUT are not both CPU or GPU arrays.") ;
    }

    numLevels = pyrLevels.getNumElements();
    if (numLevels<=0) {
        mexErrMsgTxt("LEVELS has zero elements.") ;
    }

    numROIs = dROIs.getWidth();


    if (dROIs.getWidth() != 5) {
        mexErrMsgTxt("Wrong number of elements in ROIS.") ;
    }

    if (numROIs<=0) {
        mexErrMsgTxt("ROIs has zero elements.") ;
    }


    if (verbosity > 0) {
        mexPrintf("numTotBins %d depth %d numROIs %d\n",numTotBins,data.getDepth(),numROIs);
    }
    /* Get the output geometry */
    vl::TensorShape outputShape(1, numTotBins,
                                data.getDepth(),
                                numROIs) ;

    vl::TensorShape dataShape = data.getShape();

    if(dataShape.getNumDimensions()<4) {
        dataShape.reshape(4);
    }

    /* Create output buffers */
    vl::Device deviceType = data.getDeviceType() ;
    vl::Type dataType = data.getDataType() ;
    vl::MexTensor output(context) ;
    vl::MexTensor derData(context) ;

    if (verbosity > 0) {
        vl::print("vl_nnspp: data: ", data) ;
        if (backMode) {
            vl::print("vl_nnspp: derOutput: ", derOutput) ;
            vl::print("vl_nnspp: derData: ", derData) ;
        } else {
            vl::print("vl_nnspp: output: ", output) ;
        }
    }



    if (!backMode) {
        output.initWithZeros(deviceType, dataType, outputShape) ;
    } else {
        derData.initWithZeros(deviceType, dataType, dataShape) ;
    }

    if (verbosity > 0) {
        mexPrintf("vl_spp: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::GPU) ? "GPU" : "CPU") ;
        mexPrintf("\nvl_spp: method %d numLevels %d; numROIs %d numTotBins %d\n", method, numLevels, numROIs, numTotBins);
    }

    /* -------------------------------------------------------------- */
    /*                                                    Do the work */
    /* -------------------------------------------------------------- */

    vl::Error error ;
    if (!backMode) {
        error = vl::nnspp_forward(context,
                                  output, data,
                                  method,
                                  numTotBins,
                                  pyrLevels,
                                  dROIs) ;

    } else {
        error = vl::nnspp_backward(context,
                                   derData, data, derOutput,
                                   method,
                                   numTotBins,
                                   pyrLevels,
                                   dROIs) ;
    }

    /* -------------------------------------------------------------- */
    /*                                                         Finish */
    /* -------------------------------------------------------------- */

    if (error != vl::vlSuccess) {
        mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
    }
    if (backMode) {
        out[OUT_RESULT] = derData.relinquish() ;
    } else {
        out[OUT_RESULT] = output.relinquish() ;
    }
}
